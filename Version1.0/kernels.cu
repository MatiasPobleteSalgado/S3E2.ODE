#include "hip/hip_runtime.h"
__global__ void updateV(SDL_Rect *cells, float *u1, float *u2, float *u3, float *v, int t, int *s, float *c, int nX, int nY, float dTime){
    int num = blockIdx.x * blockDim.x + threadIdx.x;
    int cT = 1;
    float rightU1, bottomU1, leftU1, topU1;
    float rightU2, bottomU2, leftU2, topU2;
    float rightU3, bottomU3, leftU3, topU3;
    float uT = u1[num], vT = v[num], dTemp, dV;
    float rightV, bottomV, leftV, topV;
    int width = cells[num].w, height = cells[num].h;
    if(s[num] != 0){
        if(s[num] == t){
            v[num] = 1000;
            return;
        }
        if((s[num] == 1) || (s[num] == 2) || (s[num] == 3)){
            v[num] = 0;
        }
    }
    if(num < nX){
        if(num == 0){
            rightU1  = u1[num + 1];
            bottomU1 = u1[num + nX];
            rightU2  = u2[num + 1];
            bottomU2 = u2[num + nX];
            rightU3  = u3[num + 1];
            bottomU3 = u3[num + nX];
            rightV  = v[num + 1];
            bottomV = v[num + nX];
            dV = cT * ((rightV - vT) / pow(width, 2)) +
                 cT * ((-vT + bottomV) / pow(height, 2));
            if(dV == 0){
                return;
            }
            dTemp = dV -
                    c[0] * ((rightU1 - uT) / pow(width, 2)) -
                    c[0] * ((-uT + bottomU1) / pow(height, 2)) -
                    c[0] * ((rightU2 - uT) / pow(width, 2)) -
                    c[0] * ((-uT + bottomU2) / pow(height, 2)) -
                    c[0] * ((rightU3 - uT) / pow(width, 2)) -
                    c[0] * ((-uT + bottomU3) / pow(height, 2));
            float newV = v[num] + dTemp * dTime;
            if(newV < 0){
                v[num] = 0.0f;
                return;
            }
            v[num] = newV;
            return;
        }
        if((num + 1) == nX){
            leftU1   = u1[num - 1];
            bottomU1 = u1[num + nX];
            leftU2   = u2[num - 1];
            bottomU2 = u2[num + nX];
            leftU3   = u3[num - 1];
            bottomU3 = u3[num + nX];
            leftV   = v[num - 1];
            bottomV = v[num + nX];
            dV = cT * ((-vT + leftV) / pow(width, 2)) + 
                 cT * ((-vT + bottomV) / pow(height, 2));
            if(dV == 0){
                return;
            }
            dTemp = dV -
                    c[0] * ((-uT + leftU1) / pow(width, 2)) - 
                    c[0] * ((-uT + bottomU1) / pow(height, 2)) -
                    c[1] * ((-uT + leftU2) / pow(width, 2)) - 
                    c[1] * ((-uT + bottomU2) / pow(height, 2)) -
                    c[2] * ((-uT + leftU3) / pow(width, 2)) - 
                    c[2] * ((-uT + bottomU3) / pow(height, 2));
            float newV = v[num] + dTemp * dTime;
            if(newV < 0){
                v[num] = 0.0f;
                return;
            }
            v[num] = newV;
            return;
        }
        rightU1  = u1[num + 1];
        leftU1   = u1[num - 1];
        bottomU1 = u1[num + nX];
        rightU2  = u2[num + 1];
        leftU2   = u2[num - 1];
        bottomU2 = u2[num + nX];
        rightU3  = u3[num + 1];
        leftU3   = u3[num - 1];
        bottomU3 = u3[num + nX];
        rightV  = v[num + 1];
        leftV   = v[num - 1];
        bottomV = v[num + nX];
        dV = cT * ((rightV - 2 * vT + leftV) / pow(width, 2)) +
             cT * ((-vT + bottomV) / pow(height, 2));
        if(dV == 0){
            return;
        }
        dTemp = dV -
                c[0] * ((rightU1 - 2 * uT + leftU1) / pow(width, 2)) -
                c[0] * ((-uT + bottomU1) / pow(height, 2)) -
                c[1] * ((rightU2 - 2 * uT + leftU2) / pow(width, 2)) -
                c[1] * ((-uT + bottomU2) / pow(height, 2)) -
                c[2] * ((rightU3 - 2 * uT + leftU3) / pow(width, 2)) -
                c[2] * ((-uT + bottomU3) / pow(height, 2)) ;
        float newV = v[num] + dTemp * dTime;
        if(newV < 0){
            v[num] = 0.0f;
            return;
        }
        v[num] = newV;
        return;
    }
    if((num + 1) > (nY * nX - nX)){
        if((num + 1) == (nY * nX - nX + 1)){
            rightU1  = u1[num + 1];
            topU1    = u1[num - nX];
            rightU2  = u2[num + 1];
            topU2    = u2[num - nX];
            rightU3  = u3[num + 1];
            topU3    = u3[num - nX];
            rightV  = v[num + 1];
            topV    = v[num - nX];
            dV = dTemp = cT * ((rightV - vT) / pow(width, 2)) + 
                    cT * ((topV - vT) / pow(height, 2));
            if(dV == 0){
                return;
            }
            dTemp = dV -
                    c[0] * ((rightU1 - uT) / pow(width, 2)) - 
                    c[0] * ((topU1 - uT) / pow(height, 2)) - 
                    c[1] * ((rightU2 - uT) / pow(width, 2)) - 
                    c[1] * ((topU2 - uT) / pow(height, 2)) -
                    c[2] * ((rightU3 - uT) / pow(width, 2)) - 
                    c[2] * ((topU3 - uT) / pow(height, 2));
            float newV = v[num] + dTemp * dTime;
            if(newV < 0){    
                v[num] = 0.0f;
                return;
            }
            v[num] = newV;
            return;
        }
        if((num + 1) == (nX * nY)){
            topU1    = u1[num - nX];
            leftU1   = u1[num - 1];
            topU2    = u2[num - nX];
            leftU2   = u2[num - 1];
            topU3    = u3[num - nX];
            leftU3   = u3[num - 1];
            leftV   = v[num - 1];
            topV    = v[num - nX];
            dV = cT * ((-vT + leftV) / pow(width, 2)) + 
                 cT * ((topV - vT) / pow(height, 2));
            if(dV == 0){
                return;
            }
            dTemp = dV -
                    c[0] * ((-uT + leftU1) / pow(width, 2)) - 
                    c[0] * ((topU1 - uT) / pow(height, 2)) -
                    c[1] * ((-uT + leftU2) / pow(width, 2)) - 
                    c[1] * ((topU2 - uT) / pow(height, 2)) -
                    c[2] * ((-uT + leftU3) / pow(width, 2)) - 
                    c[2] * ((topU3 - uT) / pow(height, 2));
            float newV = v[num] + dTemp * dTime;
            if(newV < 0){    
                v[num] = 0.0f;
                return;
            }
            v[num] = newV;
            return;
        }
        topU1    = u1[num - nX];
        leftU1   = u1[num - 1];
        rightU1  = u1[num + 1];
        topU2    = u2[num - nX];
        leftU2   = u2[num - 1];
        rightU2  = u2[num + 1];
        topU3    = u3[num - nX];
        leftU3   = u3[num - 1];
        rightU3  = u3[num + 1];
        rightV  = v[num + 1];
        leftV   = v[num - 1];
        topV    = v[num - nX];
        dV = cT * ((rightV - 2 * vT + leftV) / pow(width, 2)) + 
             cT * ((topV - vT) / pow(height, 2));
        if(dV == 0){
            return;
        }
        dTemp = dV -
                c[0] * ((rightU1 - 2 * uT + leftU1) / pow(width, 2)) - 
                c[0] * ((topU1 - uT) / pow(height, 2)) - 
                c[1] * ((rightU2 - 2 * uT + leftU2) / pow(width, 2)) - 
                c[1] * ((topU2 - uT) / pow(height, 2)) -
                c[2] * ((rightU3 - 2 * uT + leftU3) / pow(width, 2)) - 
                c[2] * ((topU3 - uT) / pow(height, 2));
        float newV = v[num] + dTemp * dTime;
        if(newV < 0){
            v[num] = 0.0f;
            return;
        }
        v[num] = newV;
        return;
    }
    if(((num + 1) % nX) == 1){
        topU1    = u1[num - nX];
        rightU1  = u1[num + 1];
        bottomU1 = u1[num + nX];
        topU2    = u2[num - nX];
        rightU2  = u2[num + 1];
        bottomU2 = u2[num + nX];
        topU3    = u3[num - nX];
        rightU3  = u3[num + 1];
        bottomU3 = u3[num + nX];
        rightV  = v[num + 1];
        topV    = v[num - nX];
        bottomV = v[num + nX];
        dV = cT * ((rightV - vT) / pow(width, 2)) + 
             cT * ((topV - 2 * vT + bottomV) / pow(height, 2));
        if(dV == 0){
            return;
        }
        dTemp = dV -
                c[0] * ((rightU1 - uT) / pow(width, 2)) -
                c[0] * ((topU1 - 2 * uT + bottomU1) / pow(height, 2)) -
                c[1] * ((rightU2 - uT) / pow(width, 2)) -
                c[1] * ((topU2 - 2 * uT + bottomU2) / pow(height, 2)) -
                c[2] * ((rightU3 - uT) / pow(width, 2)) -
                c[2] * ((topU3 - 2 * uT + bottomU3) / pow(height, 2));
        float newV = v[num] + dTemp * dTime;
        if(newV < 0){
            v[num] = 0.0f;
            return;
        }
        v[num] = newV;
        return;
    }
    if(((num + 1) % nX) == 0){
        leftU1   = u1[num - 1];
        topU1    = u1[num - nX];
        bottomU1 = u1[num + nX];
        leftU2   = u2[num - 1];
        topU2    = u2[num - nX];
        bottomU2 = u2[num + nX];
        leftU3   = u3[num - 1];
        topU3    = u3[num - nX];
        bottomU3 = u3[num + nX];
        leftV   = v[num - 1];
        topV    = v[num - nX];
        bottomV = v[num + nX];
        dV = cT * ((-vT + leftV) / pow(width, 2)) +
             cT * ((topV - 2 * vT + bottomV) / pow(height, 2));
        if(dV == 0){
            return;
        }
        dTemp = dV -
                c[0] * ((-uT + leftU1) / pow(width, 2)) -
                c[0] * ((topU1 - 2 * uT + bottomU1) / pow(height, 2)) -
                c[1] * ((-uT + leftU2) / pow(width, 2)) -
                c[1] * ((topU2 - 2 * uT + bottomU2) / pow(height, 2)) -
                c[2] * ((-uT + leftU3) / pow(width, 2)) -
                c[2] * ((topU3 - 2 * uT + bottomU3) / pow(height, 2));
        float newV = v[num] + dTemp * dTime;
        if(newV < 0){
            v[num] = 0.0f;
            return;
        }
        v[num] = newV;
        return;
    }
    rightU1  = u1[num + 1];
    leftU1   = u1[num - 1];
    topU1    = u1[num - nX];
    bottomU1 = u1[num + nX];
    rightU2  = u2[num + 1];
    leftU2   = u2[num - 1];
    topU2    = u2[num - nX];
    bottomU2 = u2[num + nX];
    rightU3  = u3[num + 1];
    leftU3   = u3[num - 1];
    topU3    = u3[num - nX];
    bottomU3 = u3[num + nX];
    rightV  = v[num + 1];
    leftV   = v[num - 1];
    topV    = v[num - nX];
    bottomV = v[num + nX];
    dV = cT * ((rightV - 2 * vT + leftV) / pow(width, 2)) + 
         cT * ((topV - 2 * vT + bottomV) / pow(height, 2));
    if(dV == 0){
        return;
    }
    dTemp = dV -
            c[0] * ((rightU1 - 2 * uT + leftU1) / pow(width, 2)) - 
            c[0] * ((topU1 - 2 * uT + bottomU1) / pow(height, 2)) -
            c[1] * ((rightU2 - 2 * uT + leftU2) / pow(width, 2)) - 
            c[1] * ((topU2 - 2 * uT + bottomU2) / pow(height, 2)) -
            c[2] * ((rightU3 - 2 * uT + leftU3) / pow(width, 2)) - 
            c[2] * ((topU3 - 2 * uT + bottomU3) / pow(height, 2));
    float newV = v[num] + dTemp * dTime;
    if(newV < 0){
        v[num] = 0.0f;
        return;
    }
    v[num] = newV;
    return;
}

__global__ void updateU(SDL_Rect * cells, float *u, int t, int *s, float c, int nX, int nY, float dTime){
    int num = blockIdx.x * blockDim.x + threadIdx.x;
    float right, bottom, left, top, temp = u[num], dTemp;
    int width = cells[num].w, height = cells[num].h;
    if(s[num] != 0){
        if(s[num] == t){
            u[num] = 1000;
            return;
        }
    }
    if(num < nX){
        if(num == 0){
            right  = u[num + 1];
            bottom = u[num + nX];
            dTemp  = c * ((right - temp) / pow(width, 2)) + 
                     c * ((-temp + bottom) / pow(height, 2));
            u[num] = u[num] + dTemp * dTime;
            return;
        }
        if((num + 1) == nX){
            left   = u[num - 1];
            bottom = u[num + nX];
            dTemp = c * ((-temp + left) / pow(width, 2)) +
                    c * ((-temp + bottom) / pow(height, 2));
            u[num] = u[num] + dTemp * dTime;
            return;
        }
        right  = u[num + 1];
        left   = u[num - 1];
        bottom = u[num + nX];
        dTemp  = c * ((right - 2 * temp + left) / pow(width, 2)) +
                 c * ((-temp + bottom) / pow(height, 2));
        u[num] = u[num] + dTemp * dTime;
        return;
    }
    if((num + 1) > (nY * nX - nX)){
        if((num + 1) == (nY * nX - nX + 1)){
            right  = u[num + 1];
            top    = u[num - nX];
            dTemp = c * ((right - temp) / pow(width, 2)) +
                    c * ((top - temp) / pow(height, 2));
            u[num] = u[num] + dTemp * dTime;
            return;
        }
        if((num + 1) == (nX * nY)){
            top    = u[num - nX];
            left   = u[num - 1];
            dTemp = c * ((-temp + left) / pow(width, 2)) +
                    c * ((top - temp) / pow(height, 2));
            u[num] = u[num] + dTemp * dTime;
            return;
        }
        top    = u[num - nX];
        left   = u[num - 1];
        right  = u[num + 1];
        dTemp = c * ((right - 2 * temp + left) / pow(width, 2)) +\
                c * ((top - temp) / pow(height, 2));
        u[num] = u[num] + dTemp * dTime;
        return;
    }
    if(((num + 1) % nX) == 1){
        top    = u[num - nX];
        right  = u[num + 1];
        bottom = u[num + nX];
        dTemp = c * ((right - temp) / pow(width, 2)) +
                c * ((top - 2 * temp + bottom) / pow(height, 2));
        u[num] = u[num] + dTemp * dTime;
        return;
    }
    if(((num + 1) % nX) == 0){
        left   = u[num - 1];
        top    = u[num - nX];
        bottom = u[num + nX];
        dTemp = c * ((-temp + left) / pow(width, 2)) +
                c * ((top - 2 * temp + bottom) / pow(height, 2));
        u[num] = u[num] + dTemp * dTime;
        return;
    }
    right  = u[num + 1];
    left   = u[num - 1];
    top    = u[num - nX];
    bottom = u[num + nX];
    dTemp = c * ((right - 2 * temp + left) / pow(width, 2)) +
            c * ((top - 2 * temp + bottom) / pow(height, 2));
    u[num] = u[num] + dTemp * dTime;
    return;
}
