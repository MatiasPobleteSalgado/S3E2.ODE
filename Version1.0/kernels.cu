#include "hip/hip_runtime.h"
__global__ void check_capacity(float *v1, float *v2, int *s, int *cap){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(
        (s[index] == 1) ||
        (s[index] == 2) ||
        (s[index] == 3)
        ){
            if((v1[index] + v2[index]) > cap[index]){
                s[index] = 0;
            }
    }
}

__global__ void updateV(
        SDL_Rect *cells, 
        float *u1, 
        float *u2, 
        float *u3, 
        float *v, 
        int t, 
        int *s, 
        float *c, 
        float *m,
        int nX, 
        int nY, 
        float dTime
    ){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int cT = 1;
    float rightU1, bottomU1, leftU1, topU1;
    float rightU2, bottomU2, leftU2, topU2;
    float rightU3, bottomU3, leftU3, topU3;
    float 
        uT1 = u1[index], 
        uT2 = u2[index], 
        uT3 = u3[index], 
        vT = v[index], 
        dTemp, 
        dV;
    float rightV, bottomV, leftV, topV;
    int width = cells[index].w, height = cells[index].h;
    if(s[index] != 0){
        if(s[index] == t){
            v[index] = 1000;
           return;
         }
    }
    if(index < nX){
        if(index == 0){
            rightU1  = u1[index + 1];
            bottomU1 = u1[index + nX];
            rightU2  = u2[index + 1];
            bottomU2 = u2[index + nX];
            rightU3  = u3[index + 1];
            bottomU3 = u3[index + nX];
            rightV  = v[index + 1];
            bottomV = v[index + nX];
            dV = cT * ((rightV - vT) / pow(width, 2)) +
                 cT * ((-vT + bottomV) / pow(height, 2));
            if(dV == 0){
                return;
            }
            dTemp = dV -
                    c[0] * ((rightU1 - uT1) / pow(width, 2)) -
                    c[0] * ((-uT1 + bottomU1) / pow(height, 2)) -
                    c[1] * ((rightU2 - uT2) / pow(width, 2)) -
                    c[1] * ((-uT2 + bottomU2) / pow(height, 2)) -
                    c[2] * ((rightU3 - uT3) / pow(width, 2)) -
                    c[2] * ((-uT3 + bottomU3) / pow(height, 2));
            float newV = v[index] + dTemp * dTime;
            if(newV < 0){
                v[index] = 0.0f;
                return;
            }
            if((s[index] == 1) || (s[index] == 2) || (s[index] == 3)){
                m[index] = m[index] + newV;
                v[index] = 0;
                return;
            }
            v[index] = newV;
            return;
        }
        if((index + 1) == nX){
            leftU1   = u1[index - 1];
            bottomU1 = u1[index + nX];
            leftU2   = u2[index - 1];
            bottomU2 = u2[index + nX];
            leftU3   = u3[index - 1];
            bottomU3 = u3[index + nX];
            leftV   = v[index - 1];
            bottomV = v[index + nX];
            dV = cT * ((-vT + leftV) / pow(width, 2)) + 
                 cT * ((-vT + bottomV) / pow(height, 2));
            if(dV == 0){
                return;
            }
            dTemp = dV -
                    c[0] * ((-uT1 + leftU1) / pow(width, 2)) - 
                    c[0] * ((-uT1 + bottomU1) / pow(height, 2)) -
                    c[1] * ((-uT2 + leftU2) / pow(width, 2)) - 
                    c[1] * ((-uT2 + bottomU2) / pow(height, 2)) -
                    c[2] * ((-uT3 + leftU3) / pow(width, 2)) - 
                    c[2] * ((-uT3 + bottomU3) / pow(height, 2));
            float newV = v[index] + dTemp * dTime;
            if(newV < 0){
                v[index] = 0.0f;
                return;
            }
            if((s[index] == 1) || (s[index] == 2) || (s[index] == 3)){
                m[index] = m[index] + newV;
                v[index] = 0;
                return;
            }
            v[index] = newV;
            return;
        }
        rightU1  = u1[index + 1];
        leftU1   = u1[index - 1];
        bottomU1 = u1[index + nX];
        rightU2  = u2[index + 1];
        leftU2   = u2[index - 1];
        bottomU2 = u2[index + nX];
        rightU3  = u3[index + 1];
        leftU3   = u3[index - 1];
        bottomU3 = u3[index + nX];
        rightV  = v[index + 1];
        leftV   = v[index - 1];
        bottomV = v[index + nX];
        dV = cT * ((rightV - 2 * vT + leftV) / pow(width, 2)) +
             cT * ((-vT + bottomV) / pow(height, 2));
        if(dV == 0){
            return;
        }
        dTemp = dV -
                c[0] * ((rightU1 - 2 * uT1 + leftU1) / pow(width, 2)) -
                c[0] * ((-uT1 + bottomU1) / pow(height, 2)) -
                c[1] * ((rightU2 - 2 * uT2 + leftU2) / pow(width, 2)) -
                c[1] * ((-uT2 + bottomU2) / pow(height, 2)) -
                c[2] * ((rightU3 - 2 * uT3 + leftU3) / pow(width, 2)) -
                c[2] * ((-uT3 + bottomU3) / pow(height, 2)) ;
        float newV = v[index] + dTemp * dTime;
        if(newV < 0){
            v[index] = 0.0f;
            return;
        }
        if((s[index] == 1) || (s[index] == 2) || (s[index] == 3)){
            m[index] = m[index] + newV;
            v[index] = 0;
            return;
        }
        v[index] = newV;
        return;
    }
    if((index + 1) > (nY * nX - nX)){
        if((index + 1) == (nY * nX - nX + 1)){
            rightU1  = u1[index + 1];
            topU1    = u1[index - nX];
            rightU2  = u2[index + 1];
            topU2    = u2[index - nX];
            rightU3  = u3[index + 1];
            topU3    = u3[index - nX];
            rightV  = v[index + 1];
            topV    = v[index - nX];
            dV = dTemp = cT * ((rightV - vT) / pow(width, 2)) + 
                    cT * ((topV - vT) / pow(height, 2));
            if(dV == 0){
                return;
            }
            dTemp = dV -
                    c[0] * ((rightU1 - uT1) / pow(width, 2)) - 
                    c[0] * ((topU1 - uT1) / pow(height, 2)) - 
                    c[1] * ((rightU2 - uT2) / pow(width, 2)) - 
                    c[1] * ((topU2 - uT2) / pow(height, 2)) -
                    c[2] * ((rightU3 - uT3) / pow(width, 2)) - 
                    c[2] * ((topU3 - uT3) / pow(height, 2));
            float newV = v[index] + dTemp * dTime;
            if(newV < 0){    
                v[index] = 0.0f;
                return;
            }
            if((s[index] == 1) || (s[index] == 2) || (s[index] == 3)){
                m[index] = m[index] + newV;
                v[index] = 0;
                return;
            }
            v[index] = newV;
            return;
        }
        if((index + 1) == (nX * nY)){
            topU1    = u1[index - nX];
            leftU1   = u1[index - 1];
            topU2    = u2[index - nX];
            leftU2   = u2[index - 1];
            topU3    = u3[index - nX];
            leftU3   = u3[index - 1];
            leftV   = v[index - 1];
            topV    = v[index - nX];
            dV = cT * ((-vT + leftV) / pow(width, 2)) + 
                 cT * ((topV - vT) / pow(height, 2));
            if(dV == 0){
                return;
            }
            dTemp = dV -
                    c[0] * ((-uT1 + leftU1) / pow(width, 2)) - 
                    c[0] * ((topU1 - uT1) / pow(height, 2)) -
                    c[1] * ((-uT2 + leftU2) / pow(width, 2)) - 
                    c[1] * ((topU2 - uT2) / pow(height, 2)) -
                    c[2] * ((-uT3 + leftU3) / pow(width, 2)) - 
                    c[2] * ((topU3 - uT3) / pow(height, 2));
            float newV = v[index] + dTemp * dTime;
            if(newV < 0){    
                v[index] = 0.0f;
                return;
            }
            if((s[index] == 1) || (s[index] == 2) || (s[index] == 3)){
                m[index] = m[index] + newV;
                v[index] = 0;
                return;
            }
            v[index] = newV;
            return;
        }
        topU1    = u1[index - nX];
        leftU1   = u1[index - 1];
        rightU1  = u1[index + 1];
        topU2    = u2[index - nX];
        leftU2   = u2[index - 1];
        rightU2  = u2[index + 1];
        topU3    = u3[index - nX];
        leftU3   = u3[index - 1];
        rightU3  = u3[index + 1];
        rightV  = v[index + 1];
        leftV   = v[index - 1];
        topV    = v[index - nX];
        dV = cT * ((rightV - 2 * vT + leftV) / pow(width, 2)) + 
             cT * ((topV - vT) / pow(height, 2));
        if(dV == 0){
            return;
        }
        dTemp = dV -
                c[0] * ((rightU1 - 2 * uT1 + leftU1) / pow(width, 2)) - 
                c[0] * ((topU1 - uT1) / pow(height, 2)) - 
                c[1] * ((rightU2 - 2 * uT2 + leftU2) / pow(width, 2)) - 
                c[1] * ((topU2 - uT2) / pow(height, 2)) -
                c[2] * ((rightU3 - 2 * uT3 + leftU3) / pow(width, 2)) - 
                c[2] * ((topU3 - uT3) / pow(height, 2));
        float newV = v[index] + dTemp * dTime;
        if(newV < 0){
            v[index] = 0.0f;
            return;
        }
        if((s[index] == 1) || (s[index] == 2) || (s[index] == 3)){
            m[index] = m[index] + newV;
            v[index] = 0;
            return;
        }
        v[index] = newV;
        return;
    }
    if(((index + 1) % nX) == 1){
        topU1    = u1[index - nX];
        rightU1  = u1[index + 1];
        bottomU1 = u1[index + nX];
        topU2    = u2[index - nX];
        rightU2  = u2[index + 1];
        bottomU2 = u2[index + nX];
        topU3    = u3[index - nX];
        rightU3  = u3[index + 1];
        bottomU3 = u3[index + nX];
        rightV  = v[index + 1];
        topV    = v[index - nX];
        bottomV = v[index + nX];
        dV = cT * ((rightV - vT) / pow(width, 2)) + 
             cT * ((topV - 2 * vT + bottomV) / pow(height, 2));
        if(dV == 0){
            return;
        }
        dTemp = dV -
                c[0] * ((rightU1 - uT1) / pow(width, 2)) -
                c[0] * ((topU1 - 2 * uT1 + bottomU1) / pow(height, 2)) -
                c[1] * ((rightU2 - uT2) / pow(width, 2)) -
                c[1] * ((topU2 - 2 * uT2 + bottomU2) / pow(height, 2)) -
                c[2] * ((rightU3 - uT3) / pow(width, 2)) -
                c[2] * ((topU3 - 2 * uT3 + bottomU3) / pow(height, 2));
        float newV = v[index] + dTemp * dTime;
        if(newV < 0){
            v[index] = 0.0f;
            return;
        }
        if((s[index] == 1) || (s[index] == 2) || (s[index] == 3)){
            m[index] = m[index] + newV;
            v[index] = 0;
            return;
        }
        v[index] = newV;
        return;
    }
    if(((index + 1) % nX) == 0){
        leftU1   = u1[index - 1];
        topU1    = u1[index - nX];
        bottomU1 = u1[index + nX];
        leftU2   = u2[index - 1];
        topU2    = u2[index - nX];
        bottomU2 = u2[index + nX];
        leftU3   = u3[index - 1];
        topU3    = u3[index - nX];
        bottomU3 = u3[index + nX];
        leftV   = v[index - 1];
        topV    = v[index - nX];
        bottomV = v[index + nX];
        dV = cT * ((-vT + leftV) / pow(width, 2)) +
             cT * ((topV - 2 * vT + bottomV) / pow(height, 2));
        if(dV == 0){
            return;
        }
        dTemp = dV -
                c[0] * ((-uT1 + leftU1) / pow(width, 2)) -
                c[0] * ((topU1 - 2 * uT1 + bottomU1) / pow(height, 2)) -
                c[1] * ((-uT2 + leftU2) / pow(width, 2)) -
                c[1] * ((topU2 - 2 * uT2 + bottomU2) / pow(height, 2)) -
                c[2] * ((-uT3 + leftU3) / pow(width, 2)) -
                c[2] * ((topU3 - 2 * uT3 + bottomU3) / pow(height, 2));
        float newV = v[index] + dTemp * dTime;
        if(newV < 0){
            v[index] = 0.0f;
            return;
        }
        if((s[index] == 1) || (s[index] == 2) || (s[index] == 3)){
            m[index] = m[index] + newV;
            v[index] = 0;
            return;
        }
        v[index] = newV;
        return;
    }
    rightU1  = u1[index + 1];
    leftU1   = u1[index - 1];
    topU1    = u1[index - nX];
    bottomU1 = u1[index + nX];
    rightU2  = u2[index + 1];
    leftU2   = u2[index - 1];
    topU2    = u2[index - nX];
    bottomU2 = u2[index + nX];
    rightU3  = u3[index + 1];
    leftU3   = u3[index - 1];
    topU3    = u3[index - nX];
    bottomU3 = u3[index + nX];
    rightV  = v[index + 1];
    leftV   = v[index - 1];
    topV    = v[index - nX];
    bottomV = v[index + nX];
    dV = cT * ((rightV - 2 * vT + leftV) / pow(width, 2)) + 
         cT * ((topV - 2 * vT + bottomV) / pow(height, 2));
    if(dV == 0){
        return;
    }
    dTemp = dV -
            c[0] * ((rightU1 - 2 * uT1 + leftU1) / pow(width, 2)) - 
            c[0] * ((topU1 - 2 * uT1 + bottomU1) / pow(height, 2)) -
            c[1] * ((rightU2 - 2 * uT2 + leftU2) / pow(width, 2)) - 
            c[1] * ((topU2 - 2 * uT2 + bottomU2) / pow(height, 2)) -
            c[2] * ((rightU3 - 2 * uT3 + leftU3) / pow(width, 2)) - 
            c[2] * ((topU3 - 2 * uT3 + bottomU3) / pow(height, 2));
    float newV = v[index] + dTemp * dTime;
    if(newV < 0){
        v[index] = 0.0f;
        return;
    }
    if((s[index] == 1) || (s[index] == 2) || (s[index] == 3)){
            m[index] = m[index] + newV;
            v[index] = 0;
            return;
        }
        v[index] = newV;
    return;
}

__global__ void updateU(SDL_Rect * cells, float *u, int t, int *s, float c, int nX, int nY, float dTime){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float right, bottom, left, top, temp = u[index], dTemp;
    int width = cells[index].w, height = cells[index].h;
    if(s[index] != 0){
        if(s[index] == t){
            u[index] = 1000;
            return;
        }
    }
    if(index < nX){
        if(index == 0){
            right  = u[index + 1];
            bottom = u[index + nX];
            dTemp  = c * ((right - temp) / pow(width, 2)) + 
                     c * ((-temp + bottom) / pow(height, 2));
            u[index] = u[index] + dTemp * dTime;
            return;
        }
        if((index + 1) == nX){
            left   = u[index - 1];
            bottom = u[index + nX];
            dTemp = c * ((-temp + left) / pow(width, 2)) +
                    c * ((-temp + bottom) / pow(height, 2));
            u[index] = u[index] + dTemp * dTime;
            return;
        }
        right  = u[index + 1];
        left   = u[index - 1];
        bottom = u[index + nX];
        dTemp  = c * ((right - 2 * temp + left) / pow(width, 2)) +
                 c * ((-temp + bottom) / pow(height, 2));
        u[index] = u[index] + dTemp * dTime;
        return;
    }
    if((index + 1) > (nY * nX - nX)){
        if((index + 1) == (nY * nX - nX + 1)){
            right  = u[index + 1];
            top    = u[index - nX];
            dTemp = c * ((right - temp) / pow(width, 2)) +
                    c * ((top - temp) / pow(height, 2));
            u[index] = u[index] + dTemp * dTime;
            return;
        }
        if((index + 1) == (nX * nY)){
            top    = u[index - nX];
            left   = u[index - 1];
            dTemp = c * ((-temp + left) / pow(width, 2)) +
                    c * ((top - temp) / pow(height, 2));
            u[index] = u[index] + dTemp * dTime;
            return;
        }
        top    = u[index - nX];
        left   = u[index - 1];
        right  = u[index + 1];
        dTemp = c * ((right - 2 * temp + left) / pow(width, 2)) +\
                c * ((top - temp) / pow(height, 2));
        u[index] = u[index] + dTemp * dTime;
        return;
    }
    if(((index + 1) % nX) == 1){
        top    = u[index - nX];
        right  = u[index + 1];
        bottom = u[index + nX];
        dTemp = c * ((right - temp) / pow(width, 2)) +
                c * ((top - 2 * temp + bottom) / pow(height, 2));
        u[index] = u[index] + dTemp * dTime;
        return;
    }
    if(((index + 1) % nX) == 0){
        left   = u[index - 1];
        top    = u[index - nX];
        bottom = u[index + nX];
        dTemp = c * ((-temp + left) / pow(width, 2)) +
                c * ((top - 2 * temp + bottom) / pow(height, 2));
        u[index] = u[index] + dTemp * dTime;
        return;
    }
    right  = u[index + 1];
    left   = u[index - 1];
    top    = u[index - nX];
    bottom = u[index + nX];
    dTemp = c * ((right - 2 * temp + left) / pow(width, 2)) +
            c * ((top - 2 * temp + bottom) / pow(height, 2));
    u[index] = u[index] + dTemp * dTime;
    return;
}
