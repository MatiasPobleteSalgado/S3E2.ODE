#include <SDL2/SDL.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include "kernels.cu"
#include "schoolData.cpp"
#include <vector>
#include <cxxopts.hpp>
#include <stdio.h>
#include <string.h>
#include "distributions.h"

using namespace std;

double dimX = 1024, dimY = 585;
int nX = 1024, nY = 1024, cellIndx = 0, scale = 3, MAX = 1000;

void draw(
    SDL_Renderer *renderer1,
    int cellIndx,
    float *u1,
    float *u2,
    float *u3,
    float *v1,
    float *v2,
    float zoom
    ){
    SDL_SetRenderDrawColor(renderer1, 0, 0, 0, 255);
    SDL_RenderClear(renderer1);
    int x = 0, y = 0;
    for(int i = 0; i < cellIndx; i++){
        if(x < (nX -1)){
            x++;
        }
        else{
            x = 0;
            y++;
        }
        SDL_SetRenderDrawColor(
            renderer1, 
            (u1[i] + v1[i]) / MAX * 255, 
            (u2[i] + (v1[i] + v2[i]) * 0.5) / MAX * 255, 
            (u3[i] + v2[i]) / MAX * 255, 
            255
        );
        SDL_RenderDrawPoint(renderer1, x * zoom, y * zoom);
    }
    SDL_RenderPresent(renderer1);
}

int main (int argc, char** argv){
	bool on = true;
	srand(1);
	// Model definition

    /*
	The five diffusion variables and "attraction" coeficients
	on unified memory
	The s variable in the future can represent features of the space
	like streets. Now it represents the exact position of schools and students
	and their type so the GPU can use that information
    */
    SDL_Rect *cells; // Rectangular info {x, y, width, height} 
    float *u1, *u2, *u3, *v1, *v2, *c1, *c2, *m1, *m2;
    int *s, *cap, *ids;

    hipMallocManaged(&cells, nX * nY * sizeof(SDL_Rect));
    hipMallocManaged(&u1, nX * nY * sizeof(float));
    hipMallocManaged(&u2, nX * nY * sizeof(float));
    hipMallocManaged(&u3, nX * nY * sizeof(float));

    hipMallocManaged(&v1, nX * nY * sizeof(float));
    hipMallocManaged(&v2, nX * nY * sizeof(float));
    
    hipMallocManaged(&s,  nX * nY * sizeof(int));
    hipMallocManaged(&cap,  nX * nY * sizeof(int));
    
    hipMallocManaged(&c1, 3 * sizeof(float));
    hipMallocManaged(&c2, 3 * sizeof(float));
    
    hipMallocManaged(&m1, nX * nY * sizeof(float));
    hipMallocManaged(&m2, nX * nY * sizeof(float));

    // Just assure that all memory is clean
    hipMemset(u1, 0, nX * nY * sizeof(float));
    hipMemset(u2, 0, nX * nY * sizeof(float));
    hipMemset(u3, 0, nX * nY * sizeof(float));
    hipMemset(v1, 0, nX * nY * sizeof(float));
    hipMemset(v2, 0, nX * nY * sizeof(float));
    hipMemset(s,  0, nX * nY * sizeof(int));
    hipMemset(c1, 0, 3 * sizeof(float));
    hipMemset(c2, 0, 3 * sizeof(float));

    bool gui;
    int max_iterations;
    try {
        gui = atoi(argv[1]);
        max_iterations = atoi(argv[2]);
        // Set coef values
        // Non vulnerable
        c1[0] = atof(argv[3]); // Municipal
        c1[1] = atof(argv[4]); // Subvencionado
        c1[2] = atof(argv[5]); // Privado
     
        // Vulnerable
        c2[0] = atof(argv[6]);
        c2[1] = atof(argv[7]);
        c2[2] = atof(argv[8]);
        printf(
            "Iterations= %d Coeficients= %f %f %f %f %f %f \n", 
            max_iterations,
            c1[0],
            c1[1],
            c1[2],
            c2[0],
            c2[1],
            c2[2]
        );
    } catch( std::exception& e) {
        std::cout << "Invalid input parameters \n";
        return 1;
    }


    // Generate grid according to model values
    for(int y = 0; y < nY; y++){
    	for(int x = 0; x < nX; x++){
    		cells[cellIndx].x = x * (2048 / nX) * scale;
    		cells[cellIndx].y = y * (2048 / nY) * scale;
    		cells[cellIndx].w = 2048 / nX * scale;
    		cells[cellIndx].h = 2048 / nY * scale;
    		cellIndx++;
    	}
    }

    // Read Shool data from binary file
    vector<School> schools = getShools("schoolData.bin");

    // Populate s array with school types, and cap with capacities
    ids = (int*)malloc(sizeof(int) * nX * nY);
    for(auto e: schools){
	    int ex = e.x * dimX;
		int ey = dimY - (e.y * dimY);
		int indx = dimX * ey + ex;
        indx = indx + dimX * (nY - dimY);
        ids[indx] = e.id;

        printf("%d\n", indx);
        // Verify if used space
		if(s[indx] == 0){
			s[indx] = e.type;
            cap[indx] = e.capacity * 1000;
		}
		else{
            printf("asdasd\n");
            cap[indx + 1] = e.capacity * 1000;
			s[indx + 1] = e.type;
		}
    }


    FILE *file = fopen("school_ids.bin", "wb");
    fwrite(ids, sizeof(int), nX * nY, file);
    fclose(file);

    /*
    for(int i = 0; i < 18244; i++){
        int index = rand() % (nX * nY); 
    	s[index] = 4;
        v1[index] = 10000;
    }
    for(int i = 0; i < 12999; i++){
        int index = rand() % (nX * nY);
    	s[index] = 5;
        v2[index] = 10000;
    }
    */
    

    int max_density = 1000;
    int index = 0;
    for(int y=0; y<1024; y++){
        for(int x=0; x<1024; x++){
            v1[index] = density_center[x] * density_center[1024 - y] * max_density;
            v2[index] = density_x[x] * density_y[1024 - y] * max_density;
            index++;
        }
    }
    
    
    float zoom = 1.0f;
    int iterations = 0;
    float delta_t = 0.1f;

    /*

    for(int i = 0; i < 10000; i++){
        updateU<<<nX, nY>>>(cells, u1, 1, s, 10.0, nX, nY, delta_t);
        updateU<<<nX, nY>>>(cells, u2, 2, s, 10.0, nX, nY, delta_t);
        updateU<<<nX, nY>>>(cells, u3, 3, s, 10.0, nX, nY, delta_t);
        hipDeviceSynchronize();
        if(gui){
            //draw(renderer1, cellIndx, u1, u2, u3, v1, v2, zoom);
        }
    }

    */
    SDL_Window *scr1 = NULL;
    SDL_Renderer *renderer1 = NULL;
    SDL_Event e1;
    SDL_Rect renderer1_viewport;
    if(gui){
        // Window variables
        scr1 = SDL_CreateWindow (
            "Schools Simulation", 
            SDL_WINDOWPOS_UNDEFINED,
            SDL_WINDOWPOS_UNDEFINED,
            nX,
            nY,
            SDL_WINDOW_SHOWN
        );
        renderer1 = SDL_CreateRenderer(scr1, -1, SDL_RENDERER_ACCELERATED);
        renderer1_viewport = {0, 0 , nX, nY};
        SDL_RenderSetViewport(renderer1, &renderer1_viewport);
    }

    while(iterations < max_iterations){
        iterations++;

        if(iterations == max_iterations -1){
            for(int i = 0; i < 1024 * 1024; i++){
                if(m1[i] != 0 || m2[i] != 0){
                    printf("m1=%f m2=%f\n", m1[i] / 1000, m2[i] / 1000);
                }
            }
        }

        if(iterations % 1000 == 0){
            printf("Saving iteration %d\n", iterations);
            char file_name[15];
            sprintf(file_name, "results/m1_%d.bin", iterations);

            FILE *file = fopen(file_name, "wb");
            fwrite(m1, sizeof(float), nX * nY, file);
            fclose(file);

            sprintf(file_name, "results/m2_%d.bin", iterations);
            
            FILE *file2 = fopen(file_name, "wb");
            fwrite(m2, sizeof(float), nX * nY, file2);
            fclose(file2);  
            if(gui){
                draw(renderer1, cellIndx, u1, u2, u3, v1, v2, zoom);
            }

        }

        updateU<<<nX, nY>>>(cells, u1, 1, s, 10.0, nX, nY, delta_t);
        updateU<<<nX, nY>>>(cells, u2, 2, s, 10.0, nX, nY, delta_t);
        updateU<<<nX, nY>>>(cells, u3, 3, s, 10.0, nX, nY, delta_t);
        hipDeviceSynchronize();

        updateV<<<nX, nY>>>(cells, u1, u2, u3, cap, v1, 4, s, c1, m1, m2, nX, nY, delta_t);
        updateV<<<nX, nY>>>(cells, u1, u2, u3, cap, v2, 5, s, c2, m1, m2, nX, nY, delta_t);
        hipDeviceSynchronize();

        if(gui){
	        while( SDL_PollEvent( &e1 ) != 0 ){
	            switch(e1.type){
                    case SDL_QUIT:
                        on = false;
                        break;
                }
            }
        }
	}

    hipFree(cells);
    hipFree(u1);
    hipFree(u2);
    hipFree(u3);
    hipFree(v1);
    hipFree(v2);
    hipFree(c1);
    hipFree(c2);
    hipFree(s);

    if(gui){
        SDL_DestroyWindow(scr1);
        SDL_Quit();        
    }


    return EXIT_SUCCESS;
}