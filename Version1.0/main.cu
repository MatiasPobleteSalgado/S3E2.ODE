#include <SDL2/SDL.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include "kernels.cu"
#include "schoolData.cpp"
#include <vector>
#include <cxxopts.hpp>
#include <stdio.h>
#include <string.h>

using namespace std;

int main (int argc, char** argv){
    /*
    cxxopts::Options options("MyProgram", "One line description of MyProgram");
    options.add_options(
        ("g,gui", "Enable use interface")
    );
    bool gui = false;
    try{
        auto result = options.parse(argc, argv);
        bool gui = result["gui"].as<bool>();
    }
    catch(cxxopts::OptionSpecException e){
        printf("Spec %s \n", e.what());
        return 1;
    }
    catch(cxxopts::OptionParseException e){
        printf("Parse %s \n", e.what());
        return 1;
    }
    */

    bool gui = false;
	bool on = true;
	srand(1);
	// Model definition
    double dimX = 2048, dimY = 2048;
    int nX = 1024, nY = 1024, cellIndx = 0, scale = 1, MAX = 1000;
    /*
	The five diffusion variables and "attraction" coeficients
	on unified memory
	The s variable in the future can represent features of the space
	like streets. Now it represents the exact position of schools and students
	and their type so the GPU can use that information
    */
    SDL_Rect *cells; // Rectangular info {x, y, width, height} 
    float *u1, *u2, *u3, *v1, *v2, *c1, *c2, *m1, *m2;
    int *s, *cap;
    hipMallocManaged(&cells, nX * nY * sizeof(SDL_Rect));
    hipMallocManaged(&u1, nX * nY * sizeof(float));
    hipMallocManaged(&u2, nX * nY * sizeof(float));
    hipMallocManaged(&u3, nX * nY * sizeof(float));

    hipMallocManaged(&v1, nX * nY * sizeof(float));
    hipMallocManaged(&v2, nX * nY * sizeof(float));
    
    hipMallocManaged(&s,  nX * nY * sizeof(int));
    hipMallocManaged(&cap,  nX * nY * sizeof(int));
    
    hipMallocManaged(&c1, 3 * sizeof(float));
    hipMallocManaged(&c2, 3 * sizeof(float));
    
    hipMallocManaged(&m1, nX * nY * sizeof(float));
    hipMallocManaged(&m2, nX * nY * sizeof(float));

    // Just assure that all memory is clean
    hipMemset(u1, 0, nX * nY * sizeof(float));
    hipMemset(u2, 0, nX * nY * sizeof(float));
    hipMemset(u3, 0, nX * nY * sizeof(float));
    hipMemset(v1, 0, nX * nY * sizeof(float));
    hipMemset(v2, 0, nX * nY * sizeof(float));
    hipMemset(s,  0, nX * nY * sizeof(int));
    hipMemset(c1, 0, 3 * sizeof(float));
    hipMemset(c2, 0, 3 * sizeof(float));

    // Set coef values


    // Non vulnerable
    c1[0] = -0.1f; // Municipal
    c1[1] = 5.0f;  // Subvencionado
    c1[2] = 10.0f; // Privado
 
    // Vulnerable
    c2[0] = 10.0f;
    c2[1] = 5.0f;
    c2[2] = -0.1f;

    // Generate grid according to desired values
    for(int y = 0; y < nY; y++){
    	for(int x = 0; x < nX; x++){
    		cells[cellIndx].x = x * (dimX / nX) * scale;
    		cells[cellIndx].y = y * (dimY / nY) * scale;
    		cells[cellIndx].w = dimX / nX * scale;
    		cells[cellIndx].h = dimY / nY * scale;
    		cellIndx++;
    	}
    }

    // Read Shool data from binary file
    vector<School> schools = getShools("schoolData.bin");
    // Populate s array with school types 

    for(auto e: schools){
	    int ex = e.x * nX;
		int ey = nY - (e.y * nY);
		int indx = nX * ey + ex;
        // Verify used space
		if(s[indx] == 0){
			s[indx] = e.type;
            cap[indx] = e.capacity * 1000;
		}
		else{
            cap[indx + 1] = e.capacity * 1000;
			s[indx + 1] = e.type;
		}
    }

    for(int i = 0; i < 18244; i++){
        int index = rand() % (nX * nY); 
    	s[index] = 4;
        v1[index] = 2000;
    }
    for(int i = 0; i < 12999; i++){
        int index = rand() % (nX * nY);
    	s[index] = 5;
        v2[index] = 2000;
    }


    SDL_Window *scr1 = NULL;
    SDL_Renderer *renderer1 = NULL;
    SDL_Event e1;
    SDL_Rect renderer1_viewport;
    if(gui){
	    // Window variables
	    scr1 = SDL_CreateWindow (
	        "Schools Simulation", 
	        SDL_WINDOWPOS_UNDEFINED,
	        SDL_WINDOWPOS_UNDEFINED,
	        nX,
	        nY,
	        SDL_WINDOW_SHOWN
	    );
	    renderer1 = SDL_CreateRenderer(scr1, -1, SDL_RENDERER_ACCELERATED);
        renderer1_viewport = {0, 0 , 1024, 1024};
        SDL_RenderSetViewport(renderer1, &renderer1_viewport);
	    // Window variables
        /*
	    scr2 = SDL_CreateWindow (
	        "Student Simulation", 
	        SDL_WINDOWPOS_UNDEFINED,
	        SDL_WINDOWPOS_UNDEFINED,
	        nX,
	        nY,
	        SDL_WINDOW_SHOWN
	    );
	    renderer2 = SDL_CreateRenderer(scr2, -1, SDL_RENDERER_ACCELERATED);
        */
    }
    float zoom = 1.0f;
    int iterations = 0;
    int max_iterations = 20000;

    float delta_t = 0.01;

    for(int i = 0; i < 10000; i++){
        updateU<<<nX, nY>>>(cells, u1, 1, s, 10.0, nX, nY, delta_t);
        updateU<<<nX, nY>>>(cells, u2, 2, s, 10.0, nX, nY, delta_t);
        updateU<<<nX, nY>>>(cells, u3, 3, s, 10.0, nX, nY, delta_t);
        hipDeviceSynchronize();

        if(gui){
            SDL_SetRenderDrawColor(renderer1, 0, 0, 0, 255);
            SDL_RenderClear(renderer1);
            while( SDL_PollEvent( &e1 ) != 0 ){
                switch(e1.type){
                    case SDL_QUIT:
                        on = false;
                        break;
                }
            }
            int x = 0, y = 0;
            for(int i = 0; i < cellIndx; i++){
                if(x < (nX -1)){
                    x++;
                }
                else{
                    x = 0;
                    y++;
                }
                SDL_SetRenderDrawColor(
                    renderer1, 
                    (u1[i]) / MAX * 255, 
                    (u2[i]) / MAX * 255, 
                    (u3[i]) / MAX * 255, 
                    255
                );
                SDL_RenderDrawPoint(renderer1, x * zoom, y * zoom);

            }
            SDL_RenderPresent(renderer1);
        }
    }

    while(iterations < max_iterations){
        iterations++;

        if(iterations % 1000 == 0){
            char file_name[15];
            sprintf(file_name, "results/m1_%d.bin", iterations);

            FILE *file = fopen(file_name, "wb");
            fwrite(m1, sizeof(float), nX * nY, file);
            fclose(file);

            sprintf(file_name, "results/m2_%d.bin", iterations);
            
            FILE *file2 = fopen(file_name, "wb");
            fwrite(m2, sizeof(float), nX * nY, file2);
            fclose(file2);            
        }

        updateU<<<nX, nY>>>(cells, u1, 1, s, 10.0, nX, nY, delta_t);
        updateU<<<nX, nY>>>(cells, u2, 2, s, 10.0, nX, nY, delta_t);
        updateU<<<nX, nY>>>(cells, u3, 3, s, 10.0, nX, nY, delta_t);
        hipDeviceSynchronize();

        updateV<<<nX, nY>>>(cells, u1, u2, u3, cap, v1, 4, s, c1, m1, m2, nX, nY, delta_t);
        updateV<<<nX, nY>>>(cells, u1, u2, u3, cap, v2, 5, s, c2, m2, m2, nX, nY, delta_t);
        hipDeviceSynchronize();

        if(gui){
		    SDL_SetRenderDrawColor(renderer1, 0, 0, 0, 255);
	    	SDL_RenderClear(renderer1);
	        while( SDL_PollEvent( &e1 ) != 0 ){
	            switch(e1.type){
                    case SDL_QUIT:
                        on = false;
                        break;
                }
            }
	        int x = 0, y = 0;
		    for(int i = 0; i < cellIndx; i++){
		    	if(x < (nX -1)){
		    		x++;
		    	}
		    	else{
		    		x = 0;
		    		y++;
		    	}
                SDL_SetRenderDrawColor(
                    renderer1, 
                    (u1[i] + v1[i]) / MAX * 255, 
                    (u2[i] + (v1[i] + v2[i]) * 0.5) / MAX * 255, 
                    (u3[i] + v2[i]) / MAX * 255, 
                    255
                );
                SDL_RenderDrawPoint(renderer1, x * zoom, y * zoom);

		    }
		    SDL_RenderPresent(renderer1);
        }
	}

    hipFree(cells);
    hipFree(u1);
    hipFree(u2);
    hipFree(u3);
    hipFree(v1);
    hipFree(v2);
    hipFree(c1);
    hipFree(c2);
    hipFree(s);

    if(gui){
        SDL_DestroyWindow(scr1);
        SDL_Quit();        
    }


    return EXIT_SUCCESS;
}