#include <SDL2/SDL.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include "kernels.cu"
#include "schoolData.cpp"
#include <vector>

using namespace std;

int main (int argc, char** argv){
	bool gui = true;
	bool on = true;
	srand(time(NULL));
	// Model definition
    double dimX = 2048, dimY = 2048;
    int nX = 1024, nY = 1024, cellIndx = 0, scale = 1, MAX = 1000;
    /*
	The five diffusion variables and "attraction" coeficients
	on unified memory
	The s variable in the future can represent features of the space
	like streets. Now it represents the exact position of schools and students
	and their type so the GPU can use that information
    */
    SDL_Rect *cells; // Rectangular info {x, y, width, height} 
    float *u1, *u2, *u3, *v1, *v2, *c1, *c2;
    int *s;
    hipMallocManaged(&cells, nX * nY * sizeof(SDL_Rect));
    hipMallocManaged(&u1, nX * nY * sizeof(float));
    hipMallocManaged(&u2, nX * nY * sizeof(float));
    hipMallocManaged(&u3, nX * nY * sizeof(float));
    hipMallocManaged(&v1, nX * nY * sizeof(float));
    hipMallocManaged(&v2, nX * nY * sizeof(float));
    hipMallocManaged(&s,  nX * nY * sizeof(int));
    hipMallocManaged(&c1, 3 * sizeof(float));
    hipMallocManaged(&c2, 3 * sizeof(float));

    // Just assure that all memory is clean
    hipMemset(u1, 0, nX * nY * sizeof(float));
    hipMemset(u2, 0, nX * nY * sizeof(float));
    hipMemset(u3, 0, nX * nY * sizeof(float));
    hipMemset(v1, 0, nX * nY * sizeof(float));
    hipMemset(v2, 0, nX * nY * sizeof(float));
    hipMemset(s,  0, nX * nY * sizeof(int));
    hipMemset(c1, 0, 3 * sizeof(float));
    hipMemset(c2, 0, 3 * sizeof(float));

    // Set coef values
    c1[0] = 0.5f;
    c1[1] = 0.4f; 
    c1[2] = -0.2f; 

    c2[0] = -0.1f; 
    c2[1] = 0.6f; 
    c2[2] = 0.8f; 

    // Generate grid according to desired values
    for(int y = 0; y < nY; y++){
    	for(int x = 0; x < nX; x++){
    		cells[cellIndx].x = x * (dimX / nX) * scale;
    		cells[cellIndx].y = y * (dimY / nY) * scale;
    		cells[cellIndx].w = dimX / nX * scale;
    		cells[cellIndx].h = dimY / nY * scale;
    		cellIndx++;
    	}
    }

    // Read Shool data from binary file
    vector<School> schools = getShools("schoolData.bin");
    // Populate s array with school types 
    for(auto e: schools){
	    int ex = e.x * nX;
		int ey = nY - (e.y * nY);
		int indx = nX * ey + ex;
		if(s[indx] == 0){
			s[indx] = e.type;
		}
		else{
			s[indx + 1] = e.type;
		}
    }

    for(int i = 0; i < 200; i++){
    	s[rand() % (nX * nY)] = 4;
    }
    for(int i = 0; i < 100; i++){
    	s[rand() % (nX * nY)] = 5;
    }

    SDL_Window *scr1, *scr2 = NULL;
    SDL_Renderer *renderer1, *renderer2 = NULL;
    SDL_Event e1, e2;
    if(gui){
	    // Window variables
	    scr1 = SDL_CreateWindow (
	        "Schools Simulation", 
	        SDL_WINDOWPOS_UNDEFINED,
	        SDL_WINDOWPOS_UNDEFINED,
	        nX,
	        nY,
	        SDL_WINDOW_SHOWN
	    );
	    renderer1 = SDL_CreateRenderer(scr1, -1, SDL_RENDERER_ACCELERATED);
	    // Window variables
	    scr2 = SDL_CreateWindow (
	        "Student Simulation", 
	        SDL_WINDOWPOS_UNDEFINED,
	        SDL_WINDOWPOS_UNDEFINED,
	        nX,
	        nY,
	        SDL_WINDOW_SHOWN
	    );
	    renderer2 = SDL_CreateRenderer(scr2, -1, SDL_RENDERER_ACCELERATED);
    }

    while(on){
        updateU<<<nX, nY>>>(cells, u1, 1, s, 0.25, nX, nY, 1);
        updateU<<<nX, nY>>>(cells, u2, 2, s, 0.5, nX, nY, 1);
        updateU<<<nX, nY>>>(cells, u3, 3, s, 1.0, nX, nY, 1);
        hipDeviceSynchronize();

        updateV<<<nX, nY>>>(cells, u1, u2, u3, v1, 4, s, c1, nX, nY, 1);
        updateV<<<nX, nY>>>(cells, u1, u2, u3, v2, 5, s, c2, nX, nY, 1);
        hipDeviceSynchronize();

        if(gui){
		    SDL_SetRenderDrawColor(renderer1, 255, 255, 255, 255);
	    	SDL_RenderClear(renderer1);
		    SDL_SetRenderDrawColor(renderer2, 255, 255, 255, 255);
	    	SDL_RenderClear(renderer2);
	        while( SDL_PollEvent( &e1 ) != 0 ){
	            if( e1.type == SDL_QUIT ){
	                on = false;
	            }
	        }
	        while( SDL_PollEvent( &e2 ) != 0 ){
	            if( e2.type == SDL_QUIT ){
	                on = false;
	            }
	        }
	        int x = 0, y = 0;

		    for(int i = 0; i < cellIndx; i++){
		    	if(x < (nX -1)){
		    		x++;
		    	}
		    	else{
		    		x = 0;
		    		y++;
		    	}
	            SDL_SetRenderDrawColor(renderer1, u1[i] / MAX * 255, u2[i] / MAX * 255, u3[i] / MAX * 255, 255);
		    	SDL_RenderDrawPoint(renderer1, x, y);
		    }
		    x = 0;
		    y = 0;
		    for(int i = 0; i < cellIndx; i++){
		    	if(x < (nX -1)){
		    		x++;
		    	}
		    	else{
		    		x = 0;
		    		y++;
		    	}
	            SDL_SetRenderDrawColor(renderer2, v1[i] / MAX * 255, 0, v2[i] / MAX * 255, 255);
		    	SDL_RenderDrawPoint(renderer2, x, y);
		    }
		    SDL_RenderPresent(renderer1);
		    SDL_RenderPresent(renderer2);
        }
	}

    hipFree(cells);
    hipFree(u1);
    hipFree(u2);
    hipFree(u3);
    hipFree(v1);
    hipFree(v2);
    hipFree(c1);
    hipFree(c2);
    hipFree(s);

    SDL_DestroyWindow(scr1);
    SDL_DestroyWindow(scr2);
    SDL_Quit();

    return EXIT_SUCCESS;
}