#include <SDL2/SDL.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include "kernels.cu"
#include "schoolData.cpp"
#include <vector>
#include <cxxopts.hpp>

using namespace std;

int main (int argc, char** argv){
    /*
    cxxopts::Options options("MyProgram", "One line description of MyProgram");
    options.add_options(
        ("g,gui", "Enable use interface")
    );
    bool gui = false;
    try{
        auto result = options.parse(argc, argv);
        bool gui = result["gui"].as<bool>();
    }
    catch(cxxopts::OptionSpecException e){
        printf("Spec %s \n", e.what());
        return 1;
    }
    catch(cxxopts::OptionParseException e){
        printf("Parse %s \n", e.what());
        return 1;
    }
    */

    bool gui = false;
	bool on = true;
	srand(time(NULL));
	// Model definition
    double dimX = 2048, dimY = 2048;
    int nX = 1024, nY = 1024, cellIndx = 0, scale = 1, MAX = 1000;
    /*
	The five diffusion variables and "attraction" coeficients
	on unified memory
	The s variable in the future can represent features of the space
	like streets. Now it represents the exact position of schools and students
	and their type so the GPU can use that information
    */
    SDL_Rect *cells; // Rectangular info {x, y, width, height} 
    float *u1, *u2, *u3, *v1, *v2, *c1, *c2, *m1, *m2;
    int *s;
    hipMallocManaged(&cells, nX * nY * sizeof(SDL_Rect));
    hipMallocManaged(&u1, nX * nY * sizeof(float));
    hipMallocManaged(&u2, nX * nY * sizeof(float));
    hipMallocManaged(&u3, nX * nY * sizeof(float));

    hipMallocManaged(&v1, nX * nY * sizeof(float));
    hipMallocManaged(&v2, nX * nY * sizeof(float));
    
    hipMallocManaged(&s,  nX * nY * sizeof(int));
    
    hipMallocManaged(&c1, 3 * sizeof(float));
    hipMallocManaged(&c2, 3 * sizeof(float));
    
    hipMallocManaged(&m1, 3 * sizeof(float));
    hipMallocManaged(&m2, 3 * sizeof(float));

    // Just assure that all memory is clean
    hipMemset(u1, 0, nX * nY * sizeof(float));
    hipMemset(u2, 0, nX * nY * sizeof(float));
    hipMemset(u3, 0, nX * nY * sizeof(float));
    hipMemset(v1, 0, nX * nY * sizeof(float));
    hipMemset(v2, 0, nX * nY * sizeof(float));
    hipMemset(s,  0, nX * nY * sizeof(int));
    hipMemset(c1, 0, 3 * sizeof(float));
    hipMemset(c2, 0, 3 * sizeof(float));

    // Set coef values
    c1[0] = -1.0f;
    c1[1] = 0.0f;
    c1[2] = 1.0f;

    c2[0] = 1.0f;
    c2[1] = 0.0f;
    c2[2] = -1.0f;

    // Generate grid according to desired values
    for(int y = 0; y < nY; y++){
    	for(int x = 0; x < nX; x++){
    		cells[cellIndx].x = x * (dimX / nX) * scale;
    		cells[cellIndx].y = y * (dimY / nY) * scale;
    		cells[cellIndx].w = dimX / nX * scale;
    		cells[cellIndx].h = dimY / nY * scale;
    		cellIndx++;
    	}
    }

    // Read Shool data from binary file
    vector<School> schools = getShools("schoolData.bin");
    // Populate s array with school types 

    for(auto e: schools){
	    int ex = e.x * nX;
		int ey = nY - (e.y * nY);
		int indx = nX * ey + ex;
        // Verify used space
		if(s[indx] == 0){
			s[indx] = e.type;
		}
		else{
			s[indx + 1] = e.type;
		}
    }

    for(int i = 0; i < 200; i++){
    	s[rand() % (nX * nY)] = 4;
    }
    for(int i = 0; i < 100; i++){
    	s[rand() % (nX * nY)] = 5;
    }

    /*
    s[0] = 2;
    s[105000] = 1;
    s[105000 + 1024 * 100] = 3;
    s[105000 + 1024 * 50 + 20] = 4;
    s[105000 + 1024 * 50 + -20] = 5;
    */
    SDL_Window *scr1 = NULL;
    SDL_Renderer *renderer1 = NULL;
    SDL_Event e1;
    SDL_Rect renderer1_viewport;
    if(gui){
	    // Window variables
	    scr1 = SDL_CreateWindow (
	        "Schools Simulation", 
	        SDL_WINDOWPOS_UNDEFINED,
	        SDL_WINDOWPOS_UNDEFINED,
	        nX,
	        nY,
	        SDL_WINDOW_SHOWN
	    );
	    renderer1 = SDL_CreateRenderer(scr1, -1, SDL_RENDERER_ACCELERATED);
        renderer1_viewport = {0, 0 , 1024, 1024};
        SDL_RenderSetViewport(renderer1, &renderer1_viewport);
	    // Window variables
        /*
	    scr2 = SDL_CreateWindow (
	        "Student Simulation", 
	        SDL_WINDOWPOS_UNDEFINED,
	        SDL_WINDOWPOS_UNDEFINED,
	        nX,
	        nY,
	        SDL_WINDOW_SHOWN
	    );
	    renderer2 = SDL_CreateRenderer(scr2, -1, SDL_RENDERER_ACCELERATED);
        */
    }
    float zoom = 1.0f;
    int nx = 0, ny = 0; 
    int iterations = 0;
    int max_iterations = 1000000; 
    while(iteracions < max_iterations){
        iterations++;
        /*
        printf(
            "v1: up=%f down=%f \nv2: up=%f down=%f \n", 
            v1[105000 + 1], 
            v1[105000 + 1024 * 100 + 1],
            v2[105000 - 1], 
            v2[105000 + 1024 * 100 - 1]
        );
        */
        updateU<<<nX, nY>>>(cells, u1, 1, s, 10.0, nX, nY, 0.1);
        updateU<<<nX, nY>>>(cells, u2, 2, s, 10.0, nX, nY, 0.1);
        updateU<<<nX, nY>>>(cells, u3, 3, s, 10.0, nX, nY, 0.1);
        hipDeviceSynchronize();

        updateV<<<nX, nY>>>(cells, u1, u2, u3, v1, 4, s, c1, m1, nX, nY, 0.1);
        updateV<<<nX, nY>>>(cells, u1, u2, u3, v2, 5, s, c2, m2, nX, nY, 0.1);
        hipDeviceSynchronize();

        if(gui){
		    SDL_SetRenderDrawColor(renderer1, 0, 0, 0, 255);
	    	SDL_RenderClear(renderer1);
	        while( SDL_PollEvent( &e1 ) != 0 ){
	            switch(e1.type){
                    case SDL_QUIT:
                        on = false;
                        break;
                    case SDL_MOUSEBUTTONDOWN:
                        switch(e1.button.button){
                            case SDL_BUTTON_LEFT:
                                zoom += 0.05;
                            break;
                            case SDL_BUTTON_RIGHT:
                                zoom -= 0.05;
                                if(zoom < 1){
                                    zoom = 1;
                                }
                            break;
                        }

                    break;
                }
	        }
            SDL_GetMouseState(&nx, &ny);
            renderer1_viewport.w = 1024 * zoom;
            renderer1_viewport.h = 1024 * zoom;
            renderer1_viewport.x = nx - renderer1_viewport.w / 2;
            renderer1_viewport.y = ny - renderer1_viewport.w / 2;
            SDL_RenderSetViewport(renderer1, &renderer1_viewport);
	        int x = 0, y = 0;
		    for(int i = 0; i < cellIndx; i++){
		    	if(x < (nX -1)){
		    		x++;
		    	}
		    	else{
		    		x = 0;
		    		y++;
		    	}
                /*
                SDL_SetRenderDrawColor(
                    renderer1, 
                    (u1[i] / MAX * 255) + (v1[i] / MAX * 255) + (v2[i] / MAX * 255), 
                    (u2[i] / MAX * 255) + (v2[i] / MAX * 255) + (v1[i] / MAX * 255), 
                    (u3[i] / MAX * 255) + (v1[i] / MAX * 255), 255
                );
                */
                SDL_SetRenderDrawColor(
                    renderer1, 
                    (u1[i] + v1[i]) / MAX * 255, 
                    (u2[i] + (v1[i] + v2[i]) * 0.5) / MAX * 255, 
                    (u3[i] + v2[i]) / MAX * 255, 
                    255
                );
                SDL_RenderDrawPoint(renderer1, x * zoom, y * zoom);

                /*
	            SDL_SetRenderDrawColor(renderer1, v1[i] / MAX * 255, 0, v2[i] / MAX * 255, 255);
		    	SDL_RenderDrawPoint(renderer1, x, y);
                */
		    }
		    SDL_RenderPresent(renderer1);
        }
	}



    hipFree(cells);
    hipFree(u1);
    hipFree(u2);
    hipFree(u3);
    hipFree(v1);
    hipFree(v2);
    hipFree(c1);
    hipFree(c2);
    hipFree(s);

    SDL_DestroyWindow(scr1);
    SDL_Quit();

    return EXIT_SUCCESS;
}