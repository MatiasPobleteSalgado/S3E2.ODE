#include "hip/hip_runtime.h"
#include <SDL2/SDL.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

__global__ void updateGrid(SDL_Rect * cells, float *temps, int nX, int nY, float dTime){
	//printf("%d\n", cells[threadIdx.x]);
    int num = blockIdx.x *blockDim.x + threadIdx.x;;
    float right, bottom, left, top, temp = temps[num], dTemp;
    int width = cells[num].w, height = cells[num].h;

    printf("%d\n", num);
    if(num < nX){
        if(num == 0){
            right  = temps[num + 1];
            bottom = temps[num + nX];
            dTemp  = (right - temp) / pow(width, 2) + (-temp + bottom) / pow(height, 2);
            temps[num] = temps[num] + dTemp * dTime;
            return;
        }
        if((num + 1) == nX){
            left   = temps[num - 1];
            bottom = temps[num + nX];
            dTemp = (-temp + left) / pow(width, 2) + (-temp + bottom) / pow(height, 2);
            temps[num] = temps[num] + dTemp * dTime;
            return;
        }
        right  = temps[num + 1];
        left   = temps[num - 1];
        bottom = temps[num + nX];
        dTemp  = (right - 2 * temp + left) / pow(width, 2) + (-temp + bottom) / pow(height, 2);
        temps[num] = temps[num] + dTemp * dTime;
        return;
    }
    if((num + 1) > (nY * nX - nX)){
        if((num + 1) == (nY * nX - nX + 1)){
            right  = temps[num + 1];
            top    = temps[num - nX];
            dTemp = (right - temp) / pow(width, 2) + (top - temp) / pow(height, 2);
            temps[num] = temps[num] + dTemp * dTime;
            return;
        }
        if((num + 1) == (nX * nY)){
            top    = temps[num - nX];
            left   = temps[num - 1];
            dTemp = (-temp + left) / pow(width, 2) + (top - temp) / pow(height, 2);
            temps[num] = temps[num] + dTemp * dTime;
            return;
        }
        top    = temps[num - nX];
        left   = temps[num - 1];
        right  = temps[num + 1];
        dTemp = (right - 2 * temp + left) / pow(width, 2) + (top - temp) / pow(height, 2);
        temps[num] = temps[num] + dTemp * dTime;
        return;
    }
    if(((num + 1) % nX) == 1){
        top    = temps[num - nX];
        right  = temps[num + 1];
        bottom = temps[num + nX];
        dTemp = (right - temp) / pow(width, 2) + (top - 2 * temp + bottom) / pow(height, 2);
        temps[num] = temps[num] + dTemp * dTime;
        return;
    }
    if(((num + 1) % nX) == 0){
        left   = temps[num - 1];
        top    = temps[num - nX];
        bottom = temps[num + nX];
        dTemp = (-temp + left) / pow(width, 2) + (top - 2 * temp + bottom) / pow(height, 2);
        temps[num] = temps[num] + dTemp * dTime;
        return;
    }
    right  = temps[num + 1];
    left   = temps[num - 1];
    top    = temps[num - nX];
    bottom = temps[num + nX];
    dTemp = (right - 2 * temp + left) / pow(width, 2) + (top - 2 * temp + bottom) / pow(height, 2);
    temps[num] = temps[num] + dTemp * dTime;
    return;
}

int main (int argc, char** argv){
	// Model definition
    double dimX = 32, dimY = 32;
    int nX = 128, nY = 128, cellIndx = 0, scale = 32;
    SDL_Rect *cells; 
    float *temperatures;
    hipMallocManaged(&cells, nX * nY * sizeof(SDL_Rect));
    hipMallocManaged(&temperatures, nX * nY * sizeof(float));

    for(int y = 0; y < nY; y++){
    	for(int x = 0; x < nX; x++){
    		cells[cellIndx].x = x * (dimX / nX) * scale;
    		cells[cellIndx].y = y * (dimY / nY) * scale;
    		cells[cellIndx].w = dimX / nX * scale;
    		cells[cellIndx].h = dimY / nY * scale;
    		cellIndx++;
    	}
    }

    // Window variables
    SDL_Window* scr = NULL;
    bool on = true;
    scr = SDL_CreateWindow (
        "Heat Simulation", 
        SDL_WINDOWPOS_UNDEFINED,
        SDL_WINDOWPOS_UNDEFINED,
        dimX * scale,
        dimY * scale,
        SDL_WINDOW_SHOWN
    );

    SDL_Renderer* renderer = NULL;
    renderer = SDL_CreateRenderer(scr, -1, SDL_RENDERER_ACCELERATED);

    SDL_Event e;
    //temperatures[0] = 1000;
    while(on){
        //   16384
        /*
        temperatures[0] = 1000;
        temperatures[50] = 1000;
        temperatures[500] = 1000;
        temperatures[500] = 1000;
        temperatures[10000] = 1000;
        temperatures[12000] = 1000;
        temperatures[15000] = 1000;
        */
        for(int j = 0; j < 11; j++){
            temperatures[rand() % cellIndx] = 1000;
        }

        //Handle events on queue
        SDL_SetRenderDrawColor(renderer, 255, 255, 255, 255);
    	SDL_RenderClear(renderer);
        while( SDL_PollEvent( &e ) != 0 ){
            //User requests quit
            if( e.type == SDL_QUIT ){
                on = false;
            }
        }
        updateGrid<<<nX, nY>>>(cells, temperatures, nX, nY, 1);
        hipDeviceSynchronize();
        // Set render color to blue ( rect will be rendered in this color )
	    // Render rect
	    for(int i = 0; i < cellIndx; i++){
            SDL_SetRenderDrawColor(renderer, temperatures[i] / 1000 * 250, 0, 0, 255 );
	    	SDL_RenderFillRect(renderer, &cells[i]);
	    }
	    SDL_RenderPresent(renderer);
	    // Wait for 5 sec
	    SDL_Delay(16);
    }

    hipFree(cells);
    hipFree(temperatures);

    SDL_DestroyWindow(scr);
    SDL_Quit();

    return EXIT_SUCCESS;
}